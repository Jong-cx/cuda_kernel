#include <bits/stdc++.h>
#include<hip/hip_runtime.h>



template <int blockSize>
__device__ float WarpShuffle(float sum) {
    if(blockSize>=32)sum += __shfl_down_sync(0xffffffff, sum, 16); 
    sum += __shfl_down_sync(0xffffffff, sum, 8);
    sum += __shfl_down_sync(0xffffffff, sum, 4);
    sum += __shfl_down_sync(0xffffffff, sum, 2);
    sum += __shfl_down_sync(0xffffffff, sum, 1);
    // 测试是否正常计算结果
    // if((blockIdx.x * blockSize + threadIdx.x)==0)
    // printf("%d sum is %f\n",0,sum);
    return sum;
}




template<int blockSize,int warpSize>
__global__ void reduce_get(float* data,float* result){
    //使用share memory
    float sum;
    int idx = threadIdx.x;
    int gidx = blockIdx.x * blockSize + threadIdx.x;
    sum = data[gidx];
    __syncthreads();
    //测试输入数据
    // if(gidx%256000==0)
    // printf("%d sum is %f\n",gidx/256000,sum);

    //使用warp层面的代码解决
    const int thread_idx = idx % warpSize;
    const int warp_idx = idx / warpSize;
    __shared__ float mem[blockSize/warpSize];
    //以warp作为基本单位计算所有block中的所有warp值
    //将warp中的值存入share mem
    sum = WarpShuffle<blockSize>(sum);
    mem[warp_idx] = sum;
    __syncthreads();
    //测试warp结果：
    // if(idx%32==0)
    // printf("%d sum is %f\n",gidx/256000,sum);

    //结果存入一个warp求和
    //只使用线程一做计算
    sum = (idx < blockSize / warpSize) ? mem[warp_idx] : 0.0;
    // if(gidx<16)  
    //     printf("gidx %d sum in one block is : %f",gidx,mem[idx]);
    sum = WarpShuffle<blockSize>(sum);

    if(idx == 0){
        result[blockIdx.x] = sum;
        __syncthreads();
        if(result[blockIdx.x] - 256.0 <= 1e-6 && result[blockIdx.x] -256 >= 1e-6)
        printf("index %d's result is wrong : %f\n",blockIdx.x,mem[idx]);
    }
}

template<int blockSize>
__global__ void reduce_get_final(float* data,float* result_final,int num){
    __shared__ float mem[blockSize];
    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    
    float x;
    for(int i = idx ;i < num ; i += step){
        x += data[gidx];
        // __syncthreads();
    }
    __syncthreads();
    //测试上面的数据是否正确
    // if(gidx == 0 || gidx == 150){
    //     printf("the thread cucalation is : %f\n", x);
    // }
    // if(gidx == 160 || gidx == 200){
    //     printf("the thread cucalation is : %f\n", x);
    // }
    mem[idx] = x;
    __syncthreads();
    int index;
    for(index = blockSize/2; index > 32; index >>= 1){
        mem[idx] += mem[idx + index];
        __syncthreads();
    }

    mem[idx] += mem[idx + 32]; __syncwarp();
    mem[idx] += mem[idx + 16]; __syncwarp();
    mem[idx] += mem[idx + 8]; __syncwarp();
    mem[idx] += mem[idx + 4]; __syncwarp();
    mem[idx] += mem[idx + 2]; __syncwarp();
    mem[idx] += mem[idx + 1]; __syncwarp();

    if(idx == 0){
        *result_final = mem[0];
        printf("sum is %f",mem[0]);
    }
}


bool getResult(const int N,float sum){
    int i = N;
    if((float)i == sum) return true;
    else return false;
}

int main(){
    float ms;
    const int N = 256 * 100000;
    
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,0);
    
    const int blockSize  = 256;
    const int warpSize = 32;
    int gridSize = std::min((N+255)/256,deviceProp.maxGridSize[0]);

    //申请内存
    float* data = (float*)malloc((N * sizeof(float)));
    float* d_data;
    for(int i = 0;i<N;i++){
        data[i] = 1.0;
    }
    hipMalloc((void**)&d_data,N*sizeof(float));
    hipMemcpy(d_data,data,N*sizeof(float),hipMemcpyHostToDevice);

    float* result = (float*)malloc(gridSize * sizeof(float));
    float* d_result;
    hipMalloc((void**)&d_result,(gridSize * sizeof(float)));

    float* resultFinal = (float*)malloc(sizeof(float));
    float* d_resultFinal;
    hipMalloc((void**)&d_resultFinal,(sizeof(float)));


    //运行kernel
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_get<blockSize,warpSize><<<gridSize,blockSize>>>(d_data,d_result);
    reduce_get_final<blockSize><<<1,blockSize>>>(d_result,d_resultFinal,gridSize);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms,start,stop);

    hipMemcpy(resultFinal,d_resultFinal,sizeof(float),hipMemcpyDeviceToHost);

    bool check = getResult(N,*resultFinal);

    if(check){
        std::cout<<"the result is right \n"<<std::endl;
    }
    else{
        std::cout<<"the result is wrong "<<std::endl;
        std::cout<<"now result is "<< *resultFinal<<std::endl;
    }
    std::cout<< "time cost is :"<<ms<<std::endl;

    free(result);
    hipFree(d_result);
    free(data);
    hipFree(d_data);
    free(resultFinal);
    hipFree(d_resultFinal);
}





