#include <iostream>
#include <hip/hip_runtime.h>

#include <hip/hip_fp16.h>

#define CUDA_CHECK_ERROR() {                                          \
    hipError_t err = hipGetLastError();                             \
    if (err != hipSuccess) {                                         \
        std::cerr << "CUDA error: " << hipGetErrorString(err)        \
                  << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
}

int main() {
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    std::cout << "Device: " << deviceProp.name << std::endl;
    std::cout << "Compute Capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;

    if (deviceProp.major >= 6) {
        std::cout << "This GPU supports FP16 operations." << std::endl;

        // Add your FP16 operation code here, e.g., vectorizedReadWrite with __hadd2 and __hmul2
    } else {
        std::cerr << "This GPU does not support FP16 operations." << std::endl;
        return 1;
    }

    CUDA_CHECK_ERROR();
    return 0;
}