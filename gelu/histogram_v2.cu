#include<bits/stdc++.h>
#include<hip/hip_runtime.h>


//使用share memory作为存储位置

__global__ void histgram(int *hist_data, int *bin_data,int N){
    __shared__ int mem[256];
    int idx = threadIdx.x;
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    // if(gidx == 0)
    // printf("the step is : %d",step);
    // //直接在block层面做原子加法
    // for(int i = 0;i<N; i += step){
    //     int val = hist_data[gidx];
    //     atomicAdd(&mem[val],1);
    //     __syncthreads();
    // }
    mem[idx] = 0;
    __syncthreads();
    if(gidx<N){
        int val = hist_data[gidx];
        atomicAdd(&mem[val],1);
    }
    __syncthreads();
    //在单个block中的计算结果是正确的
    // if(gidx < 256) {printf("gidx %d's result is : %d\n",gidx,mem[hist_data[gidx]]);}
    //使用计算完成的数据累加。block之间的原子加法
    //
    // if(gidx == 0)
    // atomicAdd(&bin_data[0],mem[0]);
    int x = mem[idx];
    atomicAdd(&bin_data[idx],x);
    __syncthreads();
    // if(gidx == 0) {printf("gidx %d's result is : %d\n",gidx,bin_data[0]);}
}




bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            // printf("in checkres, out[i]=%d, gt[i]=%d\n", out[i], groudtruth[i]);
            return false;
        }
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }
    // hist[255] = 257;
    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // bug1: L68的N不能传错，之前传的256，导致L19的cache[1]打印出来为0
    histgram<<<Grid, Block>>>(hist_data, bin_data,N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    // bug2: 同bug1，L67传进去的256表示两个buffer的数据量，这个必须得精确，之前传的N，尽管只打印第1个值，但依然导致L27打印出来的值为垃圾值
    bool is_right = CheckResult(bin, groudtruth, 256);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            // printf("%d ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram + shared_mem + multi_value latency = %f ms\n", milliseconds);    

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}